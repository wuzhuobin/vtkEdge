#include "hip/hip_runtime.h"

#include <cutil.h>
#include <hipfft/hipfft.h>

// Defined in vtkImageFourierFilter.h
// We re-difine it here so nvcc can compile it.
typedef struct{
    float Real;
    float Imag;
  } vtkImageComplexf;

extern "C"
void ExecuteFft(vtkImageComplexf *in, vtkImageComplexf *out, int N)
{
  if(N == 1)
    {
    *out++ = *in++;
    return;
    }

  //CUT_DEVICE_INIT();

  int mem_size = sizeof(vtkImageComplexf) * N;

  // Allocate device memory for signal
  float2 * d_signal;
  CUDA_SAFE_CALL(hipMalloc((void**)&d_signal, mem_size));
  // Copy host memory to device
  CUDA_SAFE_CALL(hipMemcpy(d_signal, in, mem_size, hipMemcpyHostToDevice));

  // CUFFT plan
  hipfftHandle plan;
  CUFFT_SAFE_CALL(hipfftPlan1d(&plan, N, HIPFFT_C2C, 1));

  // Transform signal
  CUFFT_SAFE_CALL(hipfftExecC2C(plan, (hipfftComplex *)d_signal, (hipfftComplex *)d_signal, HIPFFT_FORWARD));

  // Check if kernel execution generated and error
  CUT_CHECK_ERROR("Kernel execution failed [ ComplexPointwiseMulAndScale ]");

  // Copy device memory to host
  CUDA_SAFE_CALL(hipMemcpy(out, d_signal, mem_size, hipMemcpyDeviceToHost));

  //Destroy CUFFT context
  CUFFT_SAFE_CALL(hipfftDestroy(plan));

  // cleanup memory
  CUDA_SAFE_CALL(hipFree(d_signal));
}