#include "hip/hip_runtime.h"

#include <cutil.h>
#include <hipfft/hipfft.h>

// Defined in vtkImageFourierFilter.h
// We re-difine it here so nvcc can compile it.
typedef struct{
    float Real;
    float Imag;
  } vtkImageComplexf;


__global__ void Scale(float2 * in, int N, float scale)
{
  const int numThreads = blockDim.x * gridDim.x;
  const int threadID = blockIdx.x * blockDim.x + threadIdx.x;
  for (int i = threadID; i < N; i += numThreads)
    {
    in[i].x *= scale;
    in[i].y *= scale;
    }
}

extern "C"
void ExecuteRFft(vtkImageComplexf *in, vtkImageComplexf *out, int N)
{
  // If a given dimenion is 1 we just copy input to output. (e.q. when 
  // processing a 256 x 256 x 1 data set, we dont need to do a 3rd FFT 
  // in the Z dimension)
  if(N == 1)
    {
    *out++ = *in++;
    return;
    }

  //CUT_DEVICE_INIT();

  int mem_size = sizeof(vtkImageComplexf) * N;

  // Allocate device memory for signal
  float2 * d_signal;
  CUDA_SAFE_CALL(hipMalloc((void**)&d_signal, mem_size));
  // Copy host memory to device
  CUDA_SAFE_CALL(hipMemcpy(d_signal, in, mem_size, hipMemcpyHostToDevice));

  // CUFFT plan
  hipfftHandle plan;
  CUFFT_SAFE_CALL(hipfftPlan1d(&plan, N, HIPFFT_C2C, 1));

  // Transform signal
  CUFFT_SAFE_CALL(hipfftExecC2C(plan, (hipfftComplex *)d_signal, (hipfftComplex *)d_signal, HIPFFT_BACKWARD));

  // Normalize the result
  Scale<<<32, N>>>(d_signal, N, 1.0f/N);

  // Check if kernel execution generated and error
  CUT_CHECK_ERROR("Kernel execution failed [ ComplexPointwiseMulAndScale ]");

  // Copy device memory to host
  CUDA_SAFE_CALL(hipMemcpy(out, d_signal, mem_size, hipMemcpyDeviceToHost));

  //Destroy CUFFT context
  CUFFT_SAFE_CALL(hipfftDestroy(plan));

  // cleanup memory
  CUDA_SAFE_CALL(hipFree(d_signal));
}